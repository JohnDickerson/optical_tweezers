#include "hip/hip_runtime.h"
#ifndef _EVOLUTION_KERNEL_H_
#define _EVOLUTION_KERNEL_H_

#include <cutil_inline.h>
#include <cutil_math.h>

#include <stdio.h>

#include "constants.h"
#include "particle.h"
#include "io.h"
#include "laser.h"

// DO NOT USE RIGHT NOW DO NOT USE RIGHT NOW DO NOT USE RIGHT NOW

// Store our force data in a texture
// [y, z] -> {fy, fz}
texture<float2, 2, hipReadModeNormalizedFloat> force_tex;

__global__ void
evolve(particle* particles, laser* lasers, float3* random_numbers) 
{
	//float normalized_y = 0.0f;
	//float normalized_z = 0.0f;
	//float2 interp_force = tex2D(force_tex, normalized_y, normalized_z);
}

#endif

