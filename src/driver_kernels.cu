#include "hip/hip_runtime.h"
#ifndef DRIVER_KERNELS_CU
#define DRIVER_KERNELS_CU

#include "constants.h"
#include "defines.h"
#include "particle.h"
#include "laser.h"
#include "particle_list.h"
#include <cutil_inline.h>

//////  //  //  //  //  //  //  //////
//  REMEMBER TO STANDARDIZE UNITS!  //
//////  //  //  //  //  //  //  //////

using namespace tweezers;
using namespace std;

// Texture that stores the force data (laser)


extern "C" void evolve_particle(const int& num_particles, particle_list particles, laser* lasers, int step);
extern "C" void setup_cuda_constants();
extern "C" void to_force_texture(float2* h_force_array, hipArray* d_force_array,   hipChannelFormatDesc& channelDesc, const int& force_tex_width, const int& force_tex_height);
extern "C" void copy_random_numbers(float3* h_random_numbers, size_t nbytes);

// Texture that stores the force data (laser)
texture<float2, 2, hipReadModeElementType> force_tex;
// Precompute random numbers on host, store on device
__constant__ float3 cuda_random_numbers[RANDOM_COUNT];

// Store physical constants on device
__constant__ float cuda_float_constants[NUM_FLOAT_CONSTANTS];
__constant__ float3 cuda_float3_constants[NUM_FLOAT3_CONSTANTS];


// float3 vector add
__device__ float3
operator+(const float3 &a, const float3 &b)
{
	return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

// float3 scalar multiply
__device__ float3
operator*(const float3 &a, const float &c)
{
	return make_float3(a.x*c, a.y*c, a.z*c);
}

// evolves the laser (position, velocity, accel)
__global__ void
evolve_laser(laser* l)
{

}


__device__ float3
apply_f_ext(float3 p_pos, laser* l)
{
	// XY forces are symmetric, distance from particle -> laser
	float rel_y_pos = sqrtf( (p_pos.x - l->pos.x)*(p_pos.x - l->pos.x) + (p_pos.y - l->pos.y)*(p_pos.y - l->pos.y));
	// Z forces are not symmetric
	float rel_z_pos = p_pos.z - l->pos.z;  
	
	//printf("ry: %f, rz: %f\n", rel_y_pos*1e6, rel_z_pos*1e6);
	
	
	// We are told to ignore laser forces outside of the grid,
	// so just update the particle with gravity and buoyancy
	if(rel_y_pos < cuda_float_constants[GRID_Y_LOW] || rel_y_pos + cuda_float_constants[GRID_SPACING] > cuda_float_constants[GRID_Y_HIGH] || 
	rel_z_pos < cuda_float_constants[GRID_Z_LOW] || rel_z_pos + cuda_float_constants[GRID_SPACING] > cuda_float_constants[GRID_Z_HIGH])
	{
		return cuda_float3_constants[LANGEVIN_F_G] + cuda_float3_constants[LANGEVIN_F_B];
	}
	
	// Index into normalized texture:  x in [low, high] -> norm_x in [0,1]
	float normalized_rel_y = __fdividef((rel_y_pos - cuda_float_constants[GRID_Y_LOW]), (cuda_float_constants[GRID_Y_HIGH] - cuda_float_constants[GRID_Y_LOW]));
	float normalized_rel_z = __fdividef((rel_z_pos - cuda_float_constants[GRID_Z_LOW]), (cuda_float_constants[GRID_Z_HIGH] - cuda_float_constants[GRID_Z_LOW]));

	
	// Textures automatically bilinearly interpolate!
	float2 interp_force = tex2D(force_tex, normalized_rel_y, normalized_rel_z);
	//printf("fy: %f, fz: %f\n", interp_force.x*1e12, interp_force.y*1e12);

	// Need to move XY forces from local (laser) coordinate system to world coordinate system
	// The calculated Y force is in the direction of the laser
	float xDif = l->pos.x - p_pos.x;
	float yDif = l->pos.y - p_pos.y;
	float mag = sqrtf(xDif*xDif + yDif*yDif);

	xDif = __fdividef(xDif, mag);
	yDif = __fdividef(yDif, mag);

	float3 fInterpVec = make_float3(-xDif*interp_force.x, -yDif*interp_force.x, interp_force.y);
	
	
	// return the F_ext force vector in the paper -- this is the sum
	// of gravity (in +Z direction), buoyancy forces (in +Z direction), and
	// the trapping force estimated above
	return cuda_float3_constants[LANGEVIN_F_G] + cuda_float3_constants[LANGEVIN_F_B] + fInterpVec;
}

__global__ void
evolve_particle_kernel(particle_list particles, laser* lasers, int step,int num_particles) 
{
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x;

	if(p_idx>=num_particles)
		return;
	
	// Hack for single lasers, very few particles
	float3 p_pos = particles.pos[p_idx];
	float3 p_vel = particles.vel[p_idx];
	float3 p_acc = particles.acc[p_idx];

	laser* l = lasers;	

	// Get the new acceleration for the single particle
	// -gamma/m    * V(t)
	float c1 = -cuda_float_constants[LANGEVIN_GAMMA] * cuda_float_constants[INV_SPHERE_MASS];
	float3 first_term = p_vel * c1;

	// 1/m * sqrt(xi / timestep)    * N[0,1]
	float c2 = cuda_float_constants[INV_SPHERE_MASS] * sqrt( __fdividef(cuda_float_constants[LANGEVIN_XI_SQUARED], cuda_float_constants[TIME_STEP]) );
	float3 randf3 = cuda_random_numbers[step + p_idx]; 
	float3 second_term = randf3 * c2 * 10.0f;
	
	// sum of buoyancy, gravity, optical force
	float3 third_term = apply_f_ext(p_pos, l);
	third_term = third_term * cuda_float_constants[INV_SPHERE_MASS];
	

	// a(t+dt) = v(t+dt) - v(t) / dt = c1+c2+c3
	float3 accel = first_term + second_term + third_term;

	// Evolve particle's physics using second order Verlet velocity integrator
	p_pos = p_pos + (p_vel*cuda_float_constants[TIME_STEP]) + (p_acc*cuda_float_constants[TIME_STEP]*cuda_float_constants[TIME_STEP]*0.5f);
	p_vel = p_vel + (p_acc + accel)*0.5f*cuda_float_constants[TIME_STEP];
	p_acc = accel;

	particles.pos[p_idx] = p_pos;
	particles.vel[p_idx] = p_vel;
	particles.acc[p_idx] = p_acc;
}

__global__ void 
test_force_tex()
{
	
	float rel_y_pos = cuda_float_constants[GRID_Y_LOW];
	float rel_z_pos = cuda_float_constants[GRID_Z_LOW];
	for(; rel_y_pos < cuda_float_constants[GRID_Y_HIGH]; rel_y_pos += cuda_float_constants[GRID_SPACING])
	{
		for(; rel_z_pos < cuda_float_constants[GRID_Z_HIGH]; rel_z_pos += cuda_float_constants[GRID_SPACING])
		{
			float normalized_rel_y = __fdividef((rel_y_pos - cuda_float_constants[GRID_Y_LOW]), (cuda_float_constants[GRID_Y_HIGH] - cuda_float_constants[GRID_Y_LOW]));
		float normalized_rel_z = __fdividef((rel_z_pos - cuda_float_constants[GRID_Z_LOW]), (cuda_float_constants[GRID_Z_HIGH] - cuda_float_constants[GRID_Z_LOW]));

			float2 interp_force = tex2D(force_tex, normalized_rel_y, normalized_rel_z);

			//printf("y: %f, z: %f\n", rel_y_pos*1e6, rel_z_pos*1e6);
			//printf("fy: %f, fz: %f\n", interp_force.x*1e12, interp_force.y*1e12);
		}
	}

}

void copy_random_numbers(float3* h_random_numbers, size_t nbytes) {
  // Transfer our random numbers over to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL( cuda_random_numbers), h_random_numbers, nbytes );
}

void to_force_texture(float2* h_force_array, hipArray* d_force_array,   hipChannelFormatDesc& channelDesc, const int& force_tex_width, const int& force_tex_height) {
  int force_size = force_tex_width * force_tex_height * sizeof(float2);
  cutilSafeCall( hipMallocArray( &d_force_array, &channelDesc, force_tex_width, force_tex_height )); 
  cutilSafeCall( hipMemcpyToArray( d_force_array, 0, 0, h_force_array, force_size, hipMemcpyHostToDevice));

  force_tex.addressMode[0] = hipAddressModeClamp;
  force_tex.addressMode[1] = hipAddressModeClamp;
  force_tex.filterMode = hipFilterModeLinear;
  force_tex.normalized = true;    // texture access normalized to [0,1]

  cutilSafeCall( hipBindTextureToArray(force_tex, d_force_array, channelDesc));
}


void evolve_particle(const int& num_particles, particle_list particles, laser* lasers, int step) {
  // Determine block/grid sizes based on num particles
  // Hacked for very few particles right now
  dim3 dimBlock(   1, 1, 1  );
  dim3 dimGrid(   num_particles, 1,  1 );

  evolve_particle_kernel<<<dimGrid, dimBlock>>>(particles, lasers, step,num_particles);
}


void setup_cuda_constants()
{
	float host_float_constants[NUM_FLOAT_CONSTANTS];
	host_float_constants[TIME_STEP] = constants::time_step;
	host_float_constants[LANGEVIN_XI_SQUARED] = constants::langevin_xi_squared;
	host_float_constants[LANGEVIN_GAMMA] = constants::langevin_gamma;
	host_float_constants[INV_SPHERE_MASS] = constants::inv_sphere_mass;
	host_float_constants[GRID_Y_LOW] = constants::grid_Y_extent_low;
	host_float_constants[GRID_Y_HIGH] = constants::grid_Y_extent_high;
	host_float_constants[GRID_Z_LOW] = constants::grid_Z_extent_low;
	host_float_constants[GRID_Z_HIGH] = constants::grid_Z_extent_high;
	host_float_constants[GRID_SPACING] = constants::grid_spacing;

	hipMemcpyToSymbol(HIP_SYMBOL(cuda_float_constants), host_float_constants, sizeof(host_float_constants) );

	float3 host_float3_constants[NUM_FLOAT3_CONSTANTS];
	host_float3_constants[LANGEVIN_F_G] = constants::langevin_F_g;
	host_float3_constants[LANGEVIN_F_B] = constants::langevin_F_b;
	
	hipMemcpyToSymbol(HIP_SYMBOL(cuda_float3_constants), host_float3_constants, sizeof(host_float3_constants) );
}

#endif // DRIVER_KERNELS_CU
