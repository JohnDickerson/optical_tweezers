#include "hip/hip_runtime.h"
#ifndef EXPERIMENT_DRIVER_CU
#define EXPERIMENT_DRIVER_CU

#include <cutil.h>
#include <cutil_inline_runtime.h>

#include "random.hpp"

#include "constants.h"
#include "defines.h"
#include "particle.h"
#include "laser.h"
#include "particle_list.h"
#include "laser_list.hpp"


// Thrust
//#include <thrust/random/linear_congruential_engine.h>
//#include <thrust/random/normal_distribution.h>


//////  //  //  //  //  //  //  //////
//  REMEMBER TO STANDARDIZE UNITS!  //
//////  //  //  //  //  //  //  //////

using namespace tweezers;
using namespace std;

// Texture that stores the force data (laser)

extern "C" void evolve_particle(dim3& dimBlock, dim3& dimGrid,
                                uint2* states,
                                const int& num_particles, 
                                particle_list particles, 
                                laser_list lasers, int total_steps);

extern "C" void evolve_particle_compute_bounds(dim3& dimBlock, dim3& dimGrid,
                                               uint2* states,
                                               const int& num_particles, 
                                               particle_list particles, 
                                laser_list lasers, int total_steps);

extern "C" void setup_cuda_constants(uint2 A, uint2 C);
extern "C" void to_force_texture(float2* h_force_array, hipArray* d_force_array,   
                                 hipChannelFormatDesc& channelDesc, const int& force_tex_width, 
                                 const int& force_tex_height);

//extern "C" void copy_random_numbers(float3* h_random_numbers, size_t nbytes);

// Texture that stores the force data (laser)
texture<float2, 2, hipReadModeElementType> force_tex;
// Precompute random numbers on host, store on device
//__constant__ float3 cuda_random_numbers[RANDOM_COUNT];

// Store physical constants on device
__constant__ float cuda_float_constants[NUM_FLOAT_CONSTANTS];
__constant__ float3 cuda_float3_constants[NUM_FLOAT3_CONSTANTS];
__constant__ uint2 cuda_uint2_constants[NUM_UINT2_CONSTANTS];


// float3 vector add
__device__ inline float3
operator+(const float3 &a, const float3 &b)
{
  return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

// float3 vector subtract
__device__ inline float3
operator-(const float3 &a, const float3 &b)
{
  return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

// float3 scalar multiply
__device__ inline float3
operator*(const float3 &a, const float &c)
{
  return make_float3(a.x*c, a.y*c, a.z*c);
}

// evolves the laser (position, velocity, accel)
__device__ inline void
evolve_laser(float3& l_pos, const float3& l_vel)
{
  l_pos = l_pos + l_vel;
}


__device__ float3
apply_f_ext(float3 p_pos, float3 l_pos, bool& active)
{
	// XY forces are symmetric, distance from particle -> laser
	//float rel_y_pos = sqrtf( (p_pos.x - l_pos.x)*(p_pos.x - l_pos.x) + (p_pos.y - l_pos.y)*(p_pos.y - l_pos.y));
	float tempA=(p_pos.x - l_pos.x);
	float tempB=(p_pos.y - l_pos.y);
	float rel_y_pos = sqrtf(tempA*tempA+tempB*tempB);
	
	// Z forces are not symmetric
	float rel_z_pos = p_pos.z - l_pos.z;  
	
	// <HACK> Ashis does this
	if (rel_z_pos == 0.0f) 
	{
		rel_z_pos = 5e-4 * 1e-6;
	}
	// </HACK>


  // We are told to ignore laser forces outside of the grid,
  // so just update the particle with gravity and buoyancy

  if(	rel_y_pos < cuda_float_constants[GRID_Y_LOW] || rel_y_pos > cuda_float_constants[GRID_Y_HIGH] || 
        rel_z_pos < cuda_float_constants[GRID_Z_LOW] || rel_z_pos > cuda_float_constants[GRID_Z_HIGH] ||
        !active ) {
      active = false;
      return make_float3(0.0f, 0.0f, 
                         cuda_float3_constants[LANGEVIN_F_G].z - cuda_float3_constants[LANGEVIN_F_B].z  * 9.81);	
	}
	
  // Index into normalized texture:  x in [low, high] -> norm_x in [0,1]
  float normalized_rel_y = __fdividef((rel_y_pos - cuda_float_constants[GRID_Y_LOW]), (cuda_float_constants[GRID_Y_HIGH] - cuda_float_constants[GRID_Y_LOW]));
  float normalized_rel_z = __fdividef((rel_z_pos - cuda_float_constants[GRID_Z_LOW]), (cuda_float_constants[GRID_Z_HIGH] - cuda_float_constants[GRID_Z_LOW]));
  
  /*
  float2 tl = tex2D(force_tex, 0.0, 0.0);
  float2 bl = tex2D(force_tex, 0.0, 1.0);
  float2 tr = tex2D(force_tex, 1.0, 0.0);
  float2 br = tex2D(force_tex, 1.0, 1.0);
  */
  //printf("top left: %f %f , bottom left: %f %f, top right: %f %f, bottom right: %f %f\n",
  //      tl.x, tl.y, bl.x, bl.y, tr.x, tr.y, br.x, br.y);

  // Textures automatically bilinearly interpolate!
  float2 interp_force = tex2D(force_tex, normalized_rel_y, normalized_rel_z);
  float2 scale_interp_force = make_float2(interp_force.x * 1e-12, interp_force.y * 1e-12); 

  //printf("interp force: %.20e %.20e %.20e %.20e\n\n", 

  /*
  printf("normalized_rel_y: %f, normalized_rel_z: %f\n",
         normalized_rel_y, normalized_rel_z);
  printf("fy: %f, fz: %f\n", dinterp_force.x*1e12, dinterp_force.y*1e12);
  */

  /*
  // Need to move XY forces from local (laser) coordinate system to world coordinate system
  // The calculated Y force is in the direction of the laser
  float mag = sqrtf(xDif*xDif + yDif*yDif);

  if (rel_y_pos > 1e-20) { 
    float inv_mag = __fdividef(1.0f, mag);
    xDif *= inv_mag;
    yDif *= inv_mag;
  } else {
    xDif = 0.0f;
    yDif = 0.0f;
  }

  float3 fInterpVec = make_float3(xDif*scale_interp_force.x, yDif*scale_interp_force.x, scale_interp_force.y);
  */
	
  // From Ashis' code
  float xDif = p_pos.x - l_pos.x;
  float yDif = p_pos.y - l_pos.y;
  float angle = atan2f(yDif, xDif);
  float xForce = scale_interp_force.x * cosf(angle);
  float yForce = scale_interp_force.x * sinf(angle);
  float zForce = scale_interp_force.y;

  if( fabsf(sqrtf(xDif*xDif + yDif*yDif)) < 1e-18 )
  {	xForce = 0; yForce = 0; }
  if( fabsf(scale_interp_force.y) < 1e-17 )
  { zForce = 0; }
  float3 fInterpVec = make_float3(xForce, yForce, zForce);


  // return the F_ext force vector in the paper -- this is the sum
  // of gravity (in +Z direction), buoyancy forces (in -Z direction), and
  // the trapping force estimated above
  active = true;
  fInterpVec.z = fInterpVec.z + (cuda_float3_constants[LANGEVIN_F_G].z - cuda_float3_constants[LANGEVIN_F_B].z) * 9.81;	
  return fInterpVec;
}


/** propagate an rand48 RNG one iteration.
    @param Xn  the current RNG state, in 2x 24-bit formg
    @param A,C the magic constants for the RNG. For striding,
    this constants have to be adapted, see the constructor
    @result    the new RNG state X(n+1)
*/
__device__ uint2 RNG_rand48_iterate_single(uint2 Xn)//, uint2 A, uint2 C)
{
  // results and Xn are 2x 24bit to handle overflows optimally, i.e.
  // in one operation.

  // the multiplication commands however give the low and hi 32 bit,
  // which have to be converted as follows:
  // 48bit in bytes = ABCD EF (space marks 32bit boundary)
  // R0             = ABC
  // R1             =    D EF

  unsigned int R0, R1;
  uint2 A = cuda_uint2_constants[DEV_A];
  uint2 C = cuda_uint2_constants[DEV_C];

  // low 24-bit multiplication
  const unsigned int lo00 = __umul24(Xn.x, A.x);
  const unsigned int hi00 = __umulhi(Xn.x, A.x);

  // 24bit distribution of 32bit multiplication results
  R0 = (lo00 & 0xFFFFFF);
  R1 = (lo00 >> 24) | (hi00 << 8);

  R0 += C.x; R1 += C.y;

  // transfer overflows
  R1 += (R0 >> 24);
  R0 &= 0xFFFFFF;

  // cross-terms, low/hi 24-bit multiplication
  R1 += __umul24(Xn.y, A.x);
  R1 += __umul24(Xn.x, A.y);

  R1 &= 0xFFFFFF;

  return make_uint2(R0, R1);
}

/************************************************
 * Box-Muller transform from Uniform [-1,1] to Normal [0,1]
 ************************************************/
__device__ void BoxMuller(float& u1, float& u2)
{
  u1 = max(u1, 1e-20);
  u2 = max(u2, 1e-20);
  float   r = sqrtf(-2.0f * logf(u1));
  float phi = 2 * M_PI * u2;
  u1 = r * __cosf(phi);
  u2 = r * __sinf(phi);
}


/** create a set of random numbers. The random numbers are generated in blocks.
    In each block, a thread calculates one random number, the first thread the
    first one, the second the second and so on.
    @param state      the current states of the RNGS, one per thread.
    @param res        where to put the generated numbers
    @param num_blocks how many random numbers each thread generates.
    The total number of random numbers is therefore
    num_blocks*nThreads.
    @param A,C        the magic constants for the iteration. They need
    to be chosen as to advance the RNG by nThreads iterations
    at once, see the constructor.
*/
__device__ void RNG_rand48_get_float(uint2& lstate, float4& t) {
  //const int nThreads = blockDim.x*gridDim.x;
  const float inv_rand_max = __fdividef(1.0f, 2147483647.0f);
  // Assumes positive even num_blocks
  t.x = float(( lstate.x >> 17 ) | ( lstate.y << 7));
  t.x *= inv_rand_max;
  lstate = RNG_rand48_iterate_single(lstate);

  t.y = float(( lstate.x >> 17 ) | ( lstate.y << 7));
  t.y *= inv_rand_max;
  lstate = RNG_rand48_iterate_single(lstate);

  t.z = float(( lstate.x >> 17 ) | ( lstate.y << 7));
  t.z *= inv_rand_max;
  lstate = RNG_rand48_iterate_single(lstate);

  t.w = float(( lstate.x >> 17 ) | ( lstate.y << 7));
  t.w *= inv_rand_max;
  lstate = RNG_rand48_iterate_single(lstate);

  BoxMuller(t.x,t.y);
  BoxMuller(t.z,t.w);
  //  printf("rand = %f, %f, %f, %f\n", t.x, t.y, t.z, t.w);
}

__global__ __device__ void
evolve_particle_kernel(uint2* states, 
                       particle_list particles, 
                       laser_list lasers, 
                       int total_steps) {
  /*
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x;
  */
  int p_idx =
    ((gridDim.x * blockIdx.y) + blockIdx.x) * (blockDim.x*blockDim.y)  +
    (blockDim.x * threadIdx.y) + threadIdx.x ;
  
  if (p_idx < *(particles.num_particles)) {

    float3 p_pos = particles.pos[p_idx];
    float3 p_vel = particles.vel[p_idx];
    float3 p_acc = particles.acc[p_idx];
    float3 l_pos = lasers.pos[p_idx];
    float3 l_vel = lasers.vel[p_idx];
    uint2 lstate = states[p_idx];
    bool active = true;


    //float c1 = -cuda_float_constants[LANGEVIN_GAMMA] * cuda_float_constants[INV_SPHERE_MASS];
    //float c2 = cuda_float_constants[INV_SPHERE_MASS] * 
    //  sqrt( cuda_float_constants[LANGEVIN_XI_SQUARED] / cuda_float_constants[TIME_STEP] );

	float const_CONST1=cuda_float_constants[CONST1];
	float const_CONST2=cuda_float_constants[CONST2];
	float const_INV_SPHERE_MASS=cuda_float_constants[INV_SPHERE_MASS];
	float const_TIME_STEP=cuda_float_constants[TIME_STEP];
	float const_TIME_STEP_HALF=const_TIME_STEP*.5;
	float const_TIME_STEP_SQ_HALF=const_TIME_STEP*const_TIME_STEP*.5;
	

    int step = 0;
    while( step < total_steps && active ) 
	{
		//for (int step = 0; step < total_steps; ++step) {      
		// -gamma/m    * V(t)
		float3 first_term = p_vel * const_CONST1;

		// Generate the next 4 random numbers into res
		float4 res;
		RNG_rand48_get_float(lstate, res);         

		// 1/m * sqrt(xi / timestep)    * N[0,1]
		float3 second_term = make_float3(res.x, res.y, res.z) * const_CONST2;

		// sum of buoyancy, gravity, optical force
		float3 third_term = apply_f_ext(p_pos, l_pos, active);
		third_term = third_term * const_INV_SPHERE_MASS;

		// a(t+dt) = v(t+dt) - v(t) / dt = c1+c2+c3
		float3 next_acc = first_term + second_term + third_term;

		// Evolve particle's physics using second order Verlet velocity integrator
		/*
		particles.pos[p_idx] = particles.pos[p_idx] +
		( particles.vel[p_idx] * cuda_float_constants[TIME_STEP] ) + 
		( particles.acc[p_idx] * cuda_float_constants[TIME_STEP] * cuda_float_constants[TIME_STEP] * 0.5f );

		particles.vel[p_idx] = particles.vel[p_idx] + 
		((particles.acc[p_idx] + next_acc) * ( 0.5f * cuda_float_constants[TIME_STEP] ));

		particles.acc[p_idx] = next_acc;
		*/

	  /*p_pos = p_pos + 
        ( p_vel * cuda_float_constants[TIME_STEP] ) + 
        ( p_acc * cuda_float_constants[TIME_STEP] * cuda_float_constants[TIME_STEP] * 0.5f );
      p_vel = p_vel + ((p_acc + next_acc) * ( 0.5f * cuda_float_constants[TIME_STEP] ));*/

		p_pos = p_pos + ( p_vel * const_TIME_STEP ) + ( p_acc * const_TIME_STEP_SQ_HALF);
		p_vel = p_vel + ((p_acc + next_acc) *const_TIME_STEP_HALF);
		p_acc = next_acc;

		evolve_laser(l_pos, l_vel);
		++step;
    }
    
    particles.pos[p_idx] = p_pos;
    particles.vel[p_idx] = p_vel;
    particles.acc[p_idx] = p_acc;
    particles.active[p_idx] = active;

    lasers.pos[p_idx] = l_pos;
    //lasers.vel[p_idx] = l_vel;    
  }
}


__global__ void
evolve_particle_compute_bounds_kernel(uint2* states, 
                                      particle_list particles, 
                                      laser_list lasers, 
                                      int total_steps) {
  /*
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x;
  */
  int p_idx =
    ((gridDim.x * blockIdx.y) + blockIdx.x) * (blockDim.x*blockDim.y)  +
    (blockDim.x * threadIdx.y) + threadIdx.x ;
  
  if (p_idx < *(particles.num_particles)) {

    float3 p_pos = particles.pos[p_idx];
    float3 p_vel = particles.vel[p_idx];
    float3 p_acc = particles.acc[p_idx];
    uint2 lstate = states[p_idx];
    bool active = true;

    // if testing for trapping conditions, store max trapping distances
    float2 max_dists = make_float2(0.0f,0.0f);

    // Hack for single lasers
    float3 l_pos = lasers.pos[p_idx];
    float3 l_vel = lasers.vel[p_idx];
    float c1 = -cuda_float_constants[LANGEVIN_GAMMA] * cuda_float_constants[INV_SPHERE_MASS];
    float c2 = cuda_float_constants[INV_SPHERE_MASS] * 
      sqrt( cuda_float_constants[LANGEVIN_XI_SQUARED] / cuda_float_constants[TIME_STEP] );

    for (int step = 0; step < total_steps; ++step)
      {      
        // -gamma/m    * V(t)
        float3 first_term = p_vel * c1;

        // Generate the next 4 random numbers into res
        float4 res;
        RNG_rand48_get_float(lstate, res);         

        // 1/m * sqrt(xi / timestep)    * N[0,1]
        float3 second_term = make_float3(res.x, res.y, res.z) * c2;

        // sum of buoyancy, gravity, optical force
        float3 third_term = apply_f_ext(p_pos, l_pos, active);
        third_term = third_term * cuda_float_constants[INV_SPHERE_MASS];

        // a(t+dt) = v(t+dt) - v(t) / dt = c1+c2+c3
        float3 next_acc = first_term + second_term + third_term;

        // Evolve particle's physics using second order Verlet velocity integrator
        p_pos = p_pos + 
          ( p_vel * cuda_float_constants[TIME_STEP] ) + 
          ( p_acc * cuda_float_constants[TIME_STEP] * cuda_float_constants[TIME_STEP] * 0.5f );
        p_vel = p_vel + ((p_acc + next_acc) * ( 0.5f * cuda_float_constants[TIME_STEP] ));
        p_acc = next_acc;

        // See if our trapped particle has strayed farther from the optical
        // trap than it has before.  If so, store radial and z delta
        float delta_y = sqrtf( (p_pos.x - l_pos.x)*(p_pos.x - l_pos.x) + (p_pos.y - l_pos.y)*(p_pos.y - l_pos.y));
        float delta_z = p_pos.z - l_pos.z;  

        evolve_laser(l_pos, l_vel);

        // delta_y is guaranteed >= 0
        if(delta_y > max_dists.x)
          max_dists.x = delta_y;
        // we only care about z below the laser; however, z-axis
        // is flipped so we keep the greater than
        if(delta_z > max_dists.y)
          max_dists.y = delta_z;
        if (!active) break;		
      }



    particles.pos[p_idx] = p_pos;
    particles.vel[p_idx] = p_vel;
    particles.acc[p_idx] = p_acc;
    
    if(!active) {
      particles.active[p_idx] = false;
    } 

    // If we are generating trapping bounds, overwrite the particle's
    // calculated position with the max_radial and max_z distances
    particles.pos[p_idx].x = 0.0f;			
    particles.pos[p_idx].y = max_dists.x;
    particles.pos[p_idx].z = max_dists.y;
  }
}



__global__ void
find_trapping_bounds(uint2* states, 
                       int num_particles, 
                       particle_list particles, 
                       laser_list lasers, 
                       int total_steps) {
  /*
	int p_idx = blockDim.x * blockIdx.x + threadIdx.x;
  */
  int p_idx =
    ((gridDim.x * blockIdx.y) + blockIdx.x) * (blockDim.x*blockDim.y)  +
    (blockDim.x * threadIdx.y) + threadIdx.x ;
  
  if (p_idx < num_particles) {

    float3 p_pos = particles.pos[p_idx];
    float3 p_vel = particles.vel[p_idx];
    float3 p_acc = particles.acc[p_idx];
    uint2 lstate = states[p_idx];
    bool active = true;

    // Hack for single lasers
    float3 l_pos = lasers.pos[p_idx];
    float3 l_vel = lasers.vel[p_idx];

    float c1 = -cuda_float_constants[LANGEVIN_GAMMA] * cuda_float_constants[INV_SPHERE_MASS];
    float c2 = cuda_float_constants[INV_SPHERE_MASS] * 
      sqrt( cuda_float_constants[LANGEVIN_XI_SQUARED] / cuda_float_constants[TIME_STEP] );

    for (int step = 0; step < total_steps; ++step)
		{      
      // -gamma/m    * V(t)
      float3 first_term = p_vel * c1;

			// Generate the next 4 random numbers into res
      float4 res;
      RNG_rand48_get_float(lstate, res);         

      // 1/m * sqrt(xi / timestep)    * N[0,1]
      float3 second_term = make_float3(res.x, res.y, res.z) * c2;

      // sum of buoyancy, gravity, optical force
      float3 third_term = apply_f_ext(p_pos, l_pos, active);
      third_term = third_term * cuda_float_constants[INV_SPHERE_MASS];

      // a(t+dt) = v(t+dt) - v(t) / dt = c1+c2+c3
      float3 next_acc = first_term + second_term + third_term;

      // Evolve particle's physics using second order Verlet velocity integrator
      p_pos = p_pos + 
        ( p_vel * cuda_float_constants[TIME_STEP] ) + 
        ( p_acc * cuda_float_constants[TIME_STEP] * cuda_float_constants[TIME_STEP] * 0.5f );
      p_vel = p_vel + ((p_acc + next_acc) * ( 0.5f * cuda_float_constants[TIME_STEP] ));
			p_acc = next_acc;

      evolve_laser(l_pos, l_vel);
      if (!active) break;		
    }
    
    particles.pos[p_idx] = p_pos;
		particles.vel[p_idx] = p_vel;
		particles.acc[p_idx] = p_acc;

    if (!active) {
      particles.active[p_idx] = false;
    } 

  }
}




__global__ void 
test_force_tex()
{
  for(float rel_y_pos = cuda_float_constants[GRID_Y_LOW];// + 0.125e-6; 
			rel_y_pos < cuda_float_constants[GRID_Y_HIGH]; 
			rel_y_pos += cuda_float_constants[GRID_SPACING])
	{
      for(float rel_z_pos = cuda_float_constants[GRID_Z_LOW];// +0.125e-6; 
					rel_z_pos < cuda_float_constants[GRID_Z_HIGH]; 
					rel_z_pos += cuda_float_constants[GRID_SPACING])
		{
           float normalized_rel_y = __fdividef((rel_y_pos - cuda_float_constants[GRID_Y_LOW]), (cuda_float_constants[GRID_Y_HIGH] - cuda_float_constants[GRID_Y_LOW]));
  float normalized_rel_z = __fdividef((rel_z_pos - cuda_float_constants[GRID_Z_LOW]), (cuda_float_constants[GRID_Z_HIGH] - cuda_float_constants[GRID_Z_LOW]));

          float2 interp_force = tex2D(force_tex, normalized_rel_y, normalized_rel_z);

					//printf("%.10f %.10f %.10f %.10f\n", rel_y_pos * 1e6, rel_z_pos * 1e6, interp_force.x, interp_force.y);
		
		}
	}

}

/*
  void copy_random_numbers(float3* h_random_numbers, size_t nbytes) {
  // Transfer our random numbers over to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL( cuda_random_numbers), h_random_numbers, nbytes );
  }
*/
void to_force_texture(float2* h_force_array, hipArray* d_force_array,   hipChannelFormatDesc& channelDesc, const int& force_tex_width, const int& force_tex_height) {
  int force_size = force_tex_width * force_tex_height * sizeof(float2);
  cutilSafeCall( hipMallocArray( &d_force_array, &channelDesc, force_tex_width, force_tex_height )); 
  cutilSafeCall( hipMemcpyToArray( d_force_array, 0, 0, h_force_array, force_size, hipMemcpyHostToDevice));

  force_tex.addressMode[0] = hipAddressModeClamp;
  force_tex.addressMode[1] = hipAddressModeClamp;
  force_tex.filterMode = hipFilterModeLinear;
  force_tex.normalized = true;    // texture access normalized to [0,1]

  cutilSafeCall( hipBindTextureToArray(force_tex, d_force_array, channelDesc));
}

void evolve_particle(dim3& dimBlock,
                     dim3& dimGrid,
                     uint2* states,
                     const int& num_particles, 
                     particle_list particles, 
                     laser_list lasers, 
                     int total_steps) {

  evolve_particle_kernel<<<dimGrid, dimBlock>>>(states, particles, lasers, total_steps);
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    cerr << hipGetErrorString(error) << "\n";
    exit(-1);
  }         
	//dim3 ones(1,1,1);
	//test_force_tex<<<ones,ones>>>();
}


void evolve_particle_compute_bounds(dim3& dimBlock,
                                    dim3& dimGrid,
                                    uint2* states,
                                    const int& num_particles, 
                                    particle_list particles, 
                                    laser_list lasers, 
                                    int total_steps) {
        
  evolve_particle_compute_bounds_kernel<<<dimGrid, dimBlock>>>(states,particles, lasers, total_steps);
  hipError_t error = hipGetLastError();
  if(error!=hipSuccess) {
    cerr << hipGetErrorString(error) << "\n";
    exit(-1);
  }         
  //dim3 ones(1,1,1);
  //test_force_tex<<<ones,ones>>>();
}

void setup_cuda_constants(uint2 A, uint2 C)
{
  float host_float_constants[NUM_FLOAT_CONSTANTS];
  host_float_constants[TIME_STEP] = constants::time_step;
  host_float_constants[LANGEVIN_XI_SQUARED] = constants::langevin_xi_squared;
  host_float_constants[LANGEVIN_GAMMA] = constants::langevin_gamma;
  host_float_constants[INV_SPHERE_MASS] = constants::inv_sphere_mass;
  host_float_constants[GRID_Y_LOW] = constants::grid_Y_extent_low;
  host_float_constants[GRID_Y_HIGH] = constants::grid_Y_extent_high;
  host_float_constants[GRID_Z_LOW] = constants::grid_Z_extent_low;
  host_float_constants[GRID_Z_HIGH] = constants::grid_Z_extent_high;
  host_float_constants[GRID_SPACING] = constants::grid_spacing;
  host_float_constants[CONST1] = constants::c1;
  host_float_constants[CONST2] = constants::c2;
  cerr << "C2 = " << host_float_constants[CONST2] << "\n";
  hipMemcpyToSymbol(HIP_SYMBOL(cuda_float_constants), host_float_constants, sizeof(host_float_constants) );

  float3 host_float3_constants[NUM_FLOAT3_CONSTANTS];
  host_float3_constants[LANGEVIN_F_G] = constants::langevin_F_g;
  host_float3_constants[LANGEVIN_F_B] = constants::langevin_F_b;
	
  hipMemcpyToSymbol(HIP_SYMBOL(cuda_float3_constants), host_float3_constants, sizeof(host_float3_constants) );

  uint2 host_uint2_constants[NUM_UINT2_CONSTANTS];
  host_uint2_constants[DEV_A] = A;
  host_uint2_constants[DEV_C] = C;
  hipMemcpyToSymbol(HIP_SYMBOL(cuda_uint2_constants), host_uint2_constants, sizeof(host_uint2_constants) );
}

#endif // EXPERIMENT_DRIVER_CU
